#include "hip/hip_runtime.h"
#include "../../include/util/math_function_ptr.h"

//#include <hip/hip_runtime.h>
//#include <>
//
//#include "hipblas.h"

#include "../../include/config.h"
#include "../../include/util/common.h"

template<typename dtype>
__global__ void gpu_minus(const dtype* a, const dtype* b, const int size, const dtype alpha, dtype* c) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		c[index] = a[index] - alpha*b[index];
	}
}

template<typename dtype>
__global__ void gpu_column_sum_plus(const dtype* a, const int row, 
	const int column, dtype* b) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < column) {
		for (int i = 0; i < row; ++i) {
			b[index] += a[i*column + index];
		}
	}
}

template<typename dtype>
__global__ void gpu_mmadd(const dtype* a, const dtype* b,
	const int size, dtype* result) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		result[index] = a[index] + b[index];
	}
}

template<typename dtype>
__global__ void gpu_gen_git_label(const dtype* a, const int size, const int classes, dtype* b) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		b[index*classes + static_cast<int>(a[index] + 0.1)] = 1.;
	}
}

template<typename dtype>
__global__ void gpu_argmax(const dtype* a, const int row, const int column, dtype* b) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < row) {
		dtype t = a[index*column];
		int m = 0;
		for (int k = 1; k < column; ++k) {
			if (t < a[index*column + k]) {
				t = a[index*column + k];
				m = k;
			}
		}
		b[index] = m;
	}
}

template<typename dtype>
__global__ void gpu_equal_count(const dtype* a, const dtype* b, const int size, int* count) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		if (a[index] == b[index]) atomicAdd(count, 1);
	}
}

namespace BigBang {
template<typename dtype>
void bigbang_gpu_minus(const dtype* a, const dtype* b, const int size, const dtype alpha, dtype* c) {
	gpu_minus << <BigBangGetBlocks(size), THREAD_MAX_NUMS >> > (a, b, size, alpha, c);
}

template void bigbang_gpu_minus<float>(const float* a, const float* b, const int size, const float alpha, float* c);
template void bigbang_gpu_minus<double>(const double* a, const double* b, const int size, const double alpha, double* c);

template<> 
void bigbang_gpu_gemm<float>(
	bool trans_a,
	bool trans_b,
	int m,
	int n,
	int k,
	const float alpha,
	const float* a,
	const float* b,
	const float beta,
	float* c) {
	const int lda = trans_a ? m : k;
	const int ldb = trans_b ? k : n;
	hipblasOperation_t op_a = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	hipblasOperation_t op_b = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	hipblasSgemm(Config::Get().CublasHandle(), op_b, op_a, n, m, k, &alpha, b, ldb,
		a, lda, &beta, c, n);
}

template<>
void bigbang_gpu_gemm<double>(
	bool trans_a,
	bool trans_b,
	int m,
	int n,
	int k,
	const double alpha,
	const double* a,
	const double* b,
	const double beta,
	double* c) { 
	const int lda = trans_a ? m : k;
	const int ldb = trans_b ? k : n;
	hipblasOperation_t op_a = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	hipblasOperation_t op_b = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	hipblasDgemm(Config::Get().CublasHandle(), op_b, op_a, n, m, k, &alpha, b, ldb,
		a, lda, &beta, c, n);
}

template<typename dtype>
void bigbang_gpu_column_sum_plus(const dtype* a, const int row, const int column, dtype* b) {
	hipMemset(b, 0, sizeof(dtype)*column);
	gpu_column_sum_plus << <BigBangGetBlocks(column), THREAD_MAX_NUMS >> > (a, row, column, b);
}
template void bigbang_gpu_column_sum_plus<float>(const float* a, const int row, const int column, float* b);
template void bigbang_gpu_column_sum_plus<double>(const double* a, const int row, const int column, double* b);


template<typename dtype>
void bigbang_gpu_mmadd(const dtype* a, const dtype* b, 
	const int size, dtype* result) {
	gpu_mmadd << <BigBangGetBlocks(size), THREAD_MAX_NUMS >> > (a, b, size, result);
}
template void bigbang_gpu_mmadd<float>(const float* a, const float* b,
	const int size, float* result);
template void bigbang_gpu_mmadd<double>(const double* a, const double* b,
	const int size, double* result);

template<typename dtype>
void bigbang_gpu_argmax(const dtype* a, const int row, const int column, dtype* b) {
	gpu_argmax << <BigBangGetBlocks(row), THREAD_MAX_NUMS >> > (a, row, column, b);
}
template void bigbang_gpu_argmax<float>(const float* a, const int row, const int column, float* b);
template void bigbang_gpu_argmax<double>(const double* a, const int row, const int column, double* b);

template<typename dtype>
void bigbang_gpu_equals_count(const dtype* a, const dtype* b, const int size, int* count) {
	gpu_equal_count << <BigBangGetBlocks(size), THREAD_MAX_NUMS >> >(a, b, size, count) ;
}
template void bigbang_gpu_equals_count<float>(const float* a, const float* b, const int size, int* count);
template void bigbang_gpu_equals_count<double>(const double* a, const double* b, const int size, int* count);

template<typename dtype>
void bigbang_gpu_gen_fit_label(const dtype* a, const int size, const int classes, dtype* b) {
	gpu_gen_git_label << <BigBangGetBlocks(size), THREAD_MAX_NUMS >> > (a, size, classes, b);
}
template void bigbang_gpu_gen_fit_label<float>(const float* a, const int size, const int classes, float* b);
template void bigbang_gpu_gen_fit_label<double>(const double* a, const int size, const int classes, double* b);

void bigbang_gpu_random_uniform(const int size, unsigned int* output) {
	hiprandGenerate(Config::Get().CurandGenerator(), output, size);
}

}
